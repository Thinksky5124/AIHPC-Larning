#include "hip/hip_runtime.h"
/*
 * @Author       : Thinksky5124
 * @Date         : 2024-04-07 15:59:01
 * @LastEditors  : Thinksky5124
 * @LastEditTime : 2024-04-21 20:54:14
 * @Description  : file content
 * @FilePath     : /AIHPC-Larning/aihpc/kernel/cuda/add/vector_add_cuda.cu
 */
#include "vector_add_cuda.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void kernel::cuda::vector_add_kernel(const float *A, const float *B,
                                    float *C, int numElements){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i] + 0.0f;
    }
}

void kernel::cuda::launch_vector_add_kernel(const torch::Tensor &a, const torch::Tensor &b, torch::Tensor &c, bool in_place){
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    if( a.numel() != b.numel() )
    {
        throw "Tensor a and b must have the same number of elements";
    }
    // Print the vector length to be used, and compute its size
    int numElements = a.numel();
    size_t size = numElements * sizeof(float);

    float* h_A = a.data_ptr<float>();
    float* h_B = b.data_ptr<float>();
    float* h_C = c.data_ptr<float>();

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector B from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vector_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
